
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>

#define BUFLEN 500*1024*1024

using namespace std;
__constant__ int thread_counts;

__device__ unsigned int rol(const unsigned int value,
                const unsigned int steps)
        {
            return ((value << steps) | (value >> (32 - steps)));
        }


__device__ void clearWBuffert(unsigned int* buffert)
        {
            for (int pos = 16; --pos >= 0;)
            {
                buffert[pos] = 0;
            }
        }

__device__ void innerHash(unsigned int* result, unsigned int* w)
        {
            unsigned int a = result[0];
            unsigned int b = result[1];
            unsigned int c = result[2];
            unsigned int d = result[3];
            unsigned int e = result[4];

            int round = 0;

            #define sha1macro(func,val) \
            { \
                const unsigned int t = rol(a, 5) + (func) + e + val + w[round]; \
                e = d; \
                d = c; \
                c = rol(b, 30); \
                b = a; \
                a = t; \
            }

            while (round < 16)
            {
                sha1macro((b & c) | (~b & d), 0x5a827999)
                ++round;
            }
            while (round < 20)
            {
                w[round] = rol((w[round - 3] ^ w[round - 8] ^ w[round - 14] ^ w[round - 16]), 1);
                sha1macro((b & c) | (~b & d), 0x5a827999)
                ++round;
            }
            while (round < 40)
            {
                w[round] = rol((w[round - 3] ^ w[round - 8] ^ w[round - 14] ^ w[round - 16]), 1);
                sha1macro(b ^ c ^ d, 0x6ed9eba1)
                ++round;
            }
            while (round < 60)
            {
                w[round] = rol((w[round - 3] ^ w[round - 8] ^ w[round - 14] ^ w[round - 16]), 1);
                sha1macro((b & c) | (b & d) | (c & d), 0x8f1bbcdc)
                ++round;
            }
            while (round < 80)
            {
                w[round] = rol((w[round - 3] ^ w[round - 8] ^ w[round - 14] ^ w[round - 16]), 1);
                sha1macro(b ^ c ^ d, 0xca62c1d6)
                ++round;
            }

            #undef sha1macro

            result[0] += a;
            result[1] += b;
            result[2] += c;
            result[3] += d;
            result[4] += e;
        }





__device__ void calc(const void* src, const int bytelength, unsigned char* hash)
    {
        // Init the result array.
        unsigned int result[5] = { 0x67452301, 0xefcdab89, 0x98badcfe, 0x10325476, 0xc3d2e1f0 };

        // Cast the void src pointer to be the byte array we can work with.
        const unsigned char* sarray = (const unsigned char*) src;

        // The reusable round buffer
        unsigned int w[80];

        // Loop through all complete 64byte blocks.
        const int endOfFullBlocks = bytelength - 64;
        int endCurrentBlock;
        int currentBlock = 0;

        while (currentBlock <= endOfFullBlocks)
        {
            endCurrentBlock = currentBlock + 64;

            // Init the round buffer with the 64 byte block data.
            for (int roundPos = 0; currentBlock < endCurrentBlock; currentBlock += 4)
            {
                // This line will swap endian on big endian and keep endian on little endian.
                w[roundPos++] = (unsigned int) sarray[currentBlock + 3]
                        | (((unsigned int) sarray[currentBlock + 2]) << 8)
                        | (((unsigned int) sarray[currentBlock + 1]) << 16)
                        | (((unsigned int) sarray[currentBlock]) << 24);
            }
            innerHash(result, w);
        }

        // Handle the last and not full 64 byte block if existing.
        endCurrentBlock = bytelength - currentBlock;
        clearWBuffert(w);
        int lastBlockBytes = 0;
        for (;lastBlockBytes < endCurrentBlock; ++lastBlockBytes)
        {
            w[lastBlockBytes >> 2] |= (unsigned int) sarray[lastBlockBytes + currentBlock] << ((3 - (lastBlockBytes & 3)) << 3);
        }
        w[lastBlockBytes >> 2] |= 0x80 << ((3 - (lastBlockBytes & 3)) << 3);
        if (endCurrentBlock >= 56)
        {
            innerHash(result, w);
            clearWBuffert(w);
        }
        w[15] = bytelength << 3;
        innerHash(result, w);

        // Store hash in result pointer, and make sure we get in in the correct order on both endian models.
        for (int hashByte = 20; --hashByte >= 0;)
        {
            hash[hashByte] = (result[hashByte >> 2] >> (((3 - hashByte) & 0x3) << 3)) & 0xff;
        }
    }


__global__
void calculateHash(unsigned char *gdata, unsigned char *hash)
{

    int i=blockIdx.x * blockDim.x + threadIdx.x,j=0;
    if(i<=thread_counts){
    calc(&gdata[i*1024*1024],1024*1024,&hash[i*20]);
    //printf("%d---- %c\n",i,gdata[i*1024*1024]);
    }
}




int readFile(char* filename, unsigned char * &buffer,int &charRead,int offset=0){

    ifstream fileObject;
    fileObject.open(filename, ios::in|ios::binary);

    if(fileObject.is_open()){

        fileObject.seekg (0, fileObject.end);
        int length = fileObject.tellg();
        fileObject.seekg (0, fileObject.beg);
        int l;
        if( offset >= length )
			return 1;
        if(length-offset >= BUFLEN){
            l =BUFLEN;
			buffer = new unsigned char[l];
        }
        else{
			l= length-offset;
            buffer = new unsigned char[l];
			buffer[0]='S';
			buffer[l-1] = 'B';
        }

        fileObject.seekg(offset);
        fileObject.read((char*)buffer,l);
        charRead = fileObject.gcount();
        fileObject.close();

        return 0;
    }

        return -1;

}


int main()
{

    unsigned char *data1;
    int charRead,offset=0,retStatus;

    while(1){

    	retStatus = readFile("/home/ajay/test2.txt",data1,charRead,offset);
        offset+= charRead;

        if (retStatus == -1 || retStatus == 1)
            break;

        unsigned char *gdata;
        unsigned char *hash;
        const int gdsize = charRead*sizeof(char);
        hipMalloc((void**)&gdata,gdsize);
        hipError_t cuda_result_code = hipGetLastError();
        if (cuda_result_code!=hipSuccess) {
           printf("gdata message: %s\n",hipGetErrorString(cuda_result_code));
        }

        int threads = charRead/(1024*1024);
        //cout<<threads<<endl;
		int blocks = 1 + threads/32;
        //cout<<blocks<<endl;
        const int hash_size = 20 * blocks * 32 * sizeof(unsigned char);
        hipMalloc((void**)&hash, hash_size);

        cuda_result_code = hipGetLastError();

        if (cuda_result_code!=hipSuccess) {
                  printf("hash message: %s\n",hipGetErrorString(cuda_result_code));
               }

        hipMemcpy( gdata, data1, gdsize, hipMemcpyHostToDevice );

        if (cuda_result_code!=hipSuccess) {
                   printf(" memcpy message: %s\n",hipGetErrorString(cuda_result_code));
                }
        hipMemcpyToSymbol(HIP_SYMBOL(thread_counts), &threads, sizeof(int));

        calculateHash<<<blocks, 32>>>(gdata,hash);
        hipDeviceSynchronize();
        if (cuda_result_code!=hipSuccess) {
                   printf(" hashcalucation message: %s\n",hipGetErrorString(cuda_result_code));
                }

        unsigned char * hash_host = new unsigned char[hash_size];
        hipMemcpy( hash_host, hash, hash_size, hipMemcpyDeviceToHost);
        hipFree( gdata );
        hipFree( hash );

        for(int i=0;i<=threads;i++)
        {
        	for(int j=0;j<20;j++)
        		printf("%02x",hash_host[i*20+j]);
        cout<<endl;
        }
        delete[] data1;
        delete[] hash_host;
        if(retStatus==1)
            break;
    }
    printf("Completed");
    return 1;
}
